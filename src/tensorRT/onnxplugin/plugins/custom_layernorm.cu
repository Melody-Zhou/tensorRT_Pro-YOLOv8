#include "hip/hip_runtime.h"
#include <NvInfer.h>
#include <NvInferPlugin.h>
#include <vector>
#include <string>
#include <assert.h>
#include <hip/hip_fp16.h>

using namespace nvinfer1;

template<typename T>
static void __global__ layernorm_kernel(const T* x, const T* weight, const T* bias, T* y, int N, int C, float epsilon);

template<>
void __global__ layernorm_kernel<float>(const float* x, const float* weight, const float* bias, float* y, int N, int C, float epsilon){
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    if(idx >= N) return;

    // x: N, C
    // y: N, C
    // weight: C
    // bias:   C
    const float* px = x + idx * C;
    float*       py = y + idx * C;

    // reduce sum
    float sq = 0.0f;
    float s  = 0.0f;
    float diver = 1.0f / C;
    for(int ic = threadIdx.x; ic < C; ic += warpSize){
        float x = px[ic];
        s += x;
        sq = fmaf(x, x * diver, sq);
    }

    for (int mask = 16; mask > 0; mask /= 2)
        s += __shfl_xor_sync(0xffffffff, s, mask);

    for (int mask = 16; mask > 0; mask /= 2)
        sq += __shfl_xor_sync(0xffffffff, sq, mask);

    float mean = s / C;
    float rstd = rsqrtf(sq - mean * mean + epsilon);
    for(int ic = threadIdx.x; ic < C; ic += warpSize) 
        py[ic] = (px[ic] - mean) * weight[ic] * rstd + bias[ic];
}

template<>
void __global__ layernorm_kernel<half>(const half* x, const half* weight, const half* bias, half* y, int N, int C, float epsilon){
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    if(idx >= N) return;

    // x: N, C
    // y: N, C
    // weight: C
    // bias:   C
    const half* px = x + idx * C;
          half* py = y + idx * C;

    // reduce sum
    float sq = 0.0f;
    float s  = 0.0f;
    float diver = 1.0f / C;
    for(int ic = threadIdx.x; ic < C; ic += warpSize){
        float x = __half2float(px[ic]);
        s += x;
        sq = fmaf(x, x * diver, sq);
    }

    for (int mask = 16; mask > 0; mask /= 2)
        s += __shfl_xor_sync(0xffffffff, s, mask);

    for (int mask = 16; mask > 0; mask /= 2)
        sq += __shfl_xor_sync(0xffffffff, sq, mask);

    float mean = s / C;
    float rstd = rsqrtf(sq - mean * mean + epsilon);
    // for(int ic = threadIdx.x; ic < C; ic += warpSize) 
        // py[ic] = __float2half((__half2float(px[ic]) - mean) * __half2float(weight[ic]) * rstd) + bias[ic];
    // ===== modify =====
    for(int ic = threadIdx.x; ic < C; ic += warpSize) {
        float px_val = __half2float(px[ic]);
        float weight_val = __half2float(weight[ic]);
        float bias_val = __half2float(bias[ic]);
        float result = (px_val - mean) * weight_val * rstd + bias_val;
        py[ic] = __float2half(result);
    }        
}

class LayerNormPlugin : public IPluginV2DynamicExt{
public:
    float epsilon;
    int axis;

    // construct by creatation
    LayerNormPlugin(float epsilon, int axis){
        this->epsilon = epsilon;
        this->axis    = axis;
    }

    // construct by deserialization
    LayerNormPlugin(const void* data, size_t size){
        const unsigned char* pdata = (const unsigned char*)data;
        this->epsilon = *(float*)pdata;  pdata += sizeof(this->epsilon);
        this->axis    = *((int*)pdata);
    }

    IPluginV2DynamicExt* clone() const noexcept override{
        return new LayerNormPlugin(this->epsilon, this->axis);
    }

    virtual DimsExprs getOutputDimensions(
        int32_t outputIndex, DimsExprs const* inputs, int32_t nbInputs, IExprBuilder& exprBuilder) noexcept{
        return inputs[0];
    }

    virtual bool supportsFormatCombination(
        int32_t pos, PluginTensorDesc const* inOut, int32_t nbInputs, int32_t nbOutputs) noexcept{
        return inOut[pos].format == TensorFormat::kLINEAR && (inOut[pos].type == DataType::kFLOAT || inOut[pos].type == DataType::kHALF) && inOut[pos].type == inOut[0].type;
    }

    virtual void configurePlugin(DynamicPluginTensorDesc const* in, int32_t nbInputs,
        DynamicPluginTensorDesc const* out, int32_t nbOutputs) noexcept{
    }

    virtual size_t getWorkspaceSize(PluginTensorDesc const* inputs, int32_t nbInputs, PluginTensorDesc const* outputs,
        int32_t nbOutputs) const noexcept{
        return 0;
    }

    virtual int32_t enqueue(PluginTensorDesc const* inputDesc, PluginTensorDesc const* outputDesc,
        void const* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept{
        
        // B, N, C
        int N = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];
        int C = inputDesc[0].dims.d[2];
        const void* x      = inputs[0];
        const void* weight = inputs[1];
        const void* bias   = inputs[2];
        void* y            = outputs[0];

        dim3 block(32, 8);
        dim3 grid(1, (N + block.y - 1) / block.y);

        if(inputDesc[0].type == DataType::kHALF){
            layernorm_kernel<half><<<grid, block, 0, stream>>>((half*)x, (half*)weight, (half*)bias, (half*)y, N, C, this->epsilon);
        }else if(inputDesc[0].type == DataType::kFLOAT){
            layernorm_kernel<float><<<grid, block, 0, stream>>>((float*)x, (float*)weight, (float*)bias, (float*)y, N, C, this->epsilon);
        }else{
            // not implemented
            return 1;
        }
        return 0;
    }

    virtual nvinfer1::DataType getOutputDataType(
        int32_t index, nvinfer1::DataType const* inputTypes, int32_t nbInputs) const noexcept{
        return inputTypes[0];
    }

    virtual int32_t initialize() noexcept{
        return 0;
    }

    virtual void terminate() noexcept{

    }

    virtual void serialize(void* buffer) const noexcept{
        unsigned char* pdata = (unsigned char*)buffer;
        *(float*)pdata = this->epsilon;  pdata += sizeof(this->epsilon);
        *(int*)pdata   = this->axis;
    }

    virtual void destroy() noexcept{

    }

    virtual void setPluginNamespace(AsciiChar const* pluginNamespace) noexcept{
    }

    virtual AsciiChar const* getPluginNamespace() const noexcept{
        return "";
    }

    virtual AsciiChar const* getPluginType() const noexcept{
        return "CustomLayerNormalization";
    }

    virtual AsciiChar const* getPluginVersion() const noexcept{
        return "1";
    }

    virtual int32_t getNbOutputs() const noexcept {
        return 1;
    }

    virtual size_t getSerializationSize() const noexcept{
        return sizeof(this->epsilon) + sizeof(this->axis);
    }
};

class LayerNormCreater : public IPluginCreator{
public:
    std::vector<PluginField> fields;
    PluginFieldCollection field_collection;
    std::string namespace_name = "ours";

    LayerNormCreater(){
        fields = {
            PluginField{"epsilon", nullptr, PluginFieldType::kFLOAT32, 1},
            PluginField{"axis",    nullptr, PluginFieldType::kINT32, 1},
        };
        field_collection.fields   = fields.data();
        field_collection.nbFields = fields.size();
    }

    virtual AsciiChar const* getPluginName() const noexcept{
        return "CustomLayerNormalization";
    }

    virtual AsciiChar const* getPluginVersion() const noexcept{
        return "1";
    }

    virtual PluginFieldCollection const* getFieldNames() noexcept{
        return &field_collection;
    }

    virtual IPluginV2* createPlugin(AsciiChar const* name, PluginFieldCollection const* fc) noexcept{
        assert(strcmp("epsilon", fc->fields[0].name) == 0);
        assert(strcmp("axis",    fc->fields[1].name) == 0);
        float epsilon = *(float*)(fc->fields[0].data);
        int axis      = *(int*)(fc->fields[1].data);
        printf("epsilon = %g, axis=%d\n", epsilon, axis);
        return new LayerNormPlugin(epsilon, axis);
    }

    virtual IPluginV2* deserializePlugin(AsciiChar const* name, void const* serialData, size_t serialLength) noexcept{
        return new LayerNormPlugin(serialData, serialLength);
    }

    virtual void setPluginNamespace(AsciiChar const* pluginNamespace) noexcept{
    }

    virtual AsciiChar const* getPluginNamespace() const noexcept{
        return "";
    }
};

REGISTER_TENSORRT_PLUGIN(LayerNormCreater);